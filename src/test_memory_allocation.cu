#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <boost/program_options.hpp>

using namespace std;
using namespace boost::program_options;

BOOST_LOG_ATTRIBUTE_KEYWORD(line_id, "LineID", unsigned int);

//////////////////////
// kernel functions //
//////////////////////



///////////
// Class //
///////////

template <typename T>
class GPUTest
{
public:
    GPUTest(int id);
    void allocate();
    void graduallyAllocate();
    void deviceInfo();
    virtual ~GPUTest();
private:
    long long int size;
    int device_id;
};


template <typename T>
GPUTest<T>::GPUTest(int id) 
{
    hipSetDevice(id);
    device_id = id;
};

template <typename T>
GPUTest<T>::~GPUTest()
{
    hipDeviceReset();
};

template <typename T>
void GPUTest<T>::allocate(void)
{

}

template <typename T>
void GPUTest<T>::graduallyAllocate(void)
{

}
template <typename T>
void GPUTest<T>::deviceInfo(void)
{
    int dev = 0, driverVersion = 0, runtimeVersion = 0;
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Device %d: \"%s\"\n", dev, deviceProp.name);
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n",
           driverVersion / 1000, (driverVersion % 100) / 10,
           runtimeVersion / 1000, (runtimeVersion % 100) / 10);
    printf("  CUDA Capability Major/Minor version number:    %d.%d\n",
           deviceProp.major, deviceProp.minor);
    printf("  Total amount of global memory:                 %.2f GBytes (%llu "
           "bytes)\n", (float)deviceProp.totalGlobalMem / pow(1024.0, 3),
           (unsigned long long)deviceProp.totalGlobalMem);
    printf("  GPU Clock rate:                                %.0f MHz (%0.2f "
           "GHz)\n", deviceProp.clockRate * 1e-3f,
           deviceProp.clockRate * 1e-6f);
    printf("  Memory Clock rate:                             %.0f Mhz\n",
           deviceProp.memoryClockRate * 1e-3f);
    printf("  Memory Bus Width:                              %d-bit\n",
           deviceProp.memoryBusWidth);

    if (deviceProp.l2CacheSize)
    {
        printf("  L2 Cache Size:                                 %d bytes\n",
               deviceProp.l2CacheSize);
    }
    printf("  Max Texture Dimension Size (x,y,z)             1D=(%d), "
           "2D=(%d,%d), 3D=(%d,%d,%d)\n", deviceProp.maxTexture1D,
           deviceProp.maxTexture2D[0], deviceProp.maxTexture2D[1],
           deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1],
           deviceProp.maxTexture3D[2]);
    printf("  Max Layered Texture Size (dim) x layers        1D=(%d) x %d, "
           "2D=(%d,%d) x %d\n", deviceProp.maxTexture1DLayered[0],
           deviceProp.maxTexture1DLayered[1], deviceProp.maxTexture2DLayered[0],
           deviceProp.maxTexture2DLayered[1],
           deviceProp.maxTexture2DLayered[2]);
    printf("  Total amount of constant memory:               %lu bytes\n",
           deviceProp.totalConstMem);
    printf("  Total amount of shared memory per block:       %lu bytes\n",
           deviceProp.sharedMemPerBlock);
    printf("  Total number of registers available per block: %d\n",
           deviceProp.regsPerBlock);
    printf("  Warp size:                                     %d\n",
           deviceProp.warpSize);
    printf("  Maximum number of threads per multiprocessor:  %d\n",
           deviceProp.maxThreadsPerMultiProcessor);
    printf("  Maximum number of threads per block:           %d\n",
           deviceProp.maxThreadsPerBlock);
    printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
           deviceProp.maxThreadsDim[0],
           deviceProp.maxThreadsDim[1],
           deviceProp.maxThreadsDim[2]);
    printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
           deviceProp.maxGridSize[0],
           deviceProp.maxGridSize[1],
           deviceProp.maxGridSize[2]);
    printf("  Maximum memory pitch:                          %lu bytes\n",
           deviceProp.memPitch);
}


///////////////////
// parse options //
///////////////////




///////////////////
// main function //
///////////////////


int main(int argc, char *argv[])
{
    int deviceId=0;

    BOOST_LOG_TRIVIAL(info) << "An informational severity message.";
    options_description options1("This programm does GPU stress test.");
    options1.add_options()
        ("help,h",    "help mesage.")
        ("deviceid,d", value<int>(),   "set DeviceId of GPU.");
        //("memory_allocation_size,s",  "set Memory allocation size (Mb).");
    
    variables_map values;
    try{
        store(parse_command_line(argc, argv, options1), values);
        notify(values);
        if (values.count("help")) {
			cout << options1 << endl;
            exit(EXIT_FAILURE);
        }
        if (!values.count("deviceid")) {
			// options_description は標準出力に投げることが出来る
			cout << options1 << endl;
            exit(EXIT_FAILURE);
		}
		if (values.count("deviceid"))
            cout << "set DeviceId: " <<  endl;
            //cout << "set DeviceId: " << values["deviceid"].as<string>() << endl;
            
    }catch(std::exception &e){
        std::cout << e.what() << std::endl;
        exit(EXIT_FAILURE);        
    }

    
    GPUTest<int> g(deviceId);
    g.deviceInfo();
    return 0;
}